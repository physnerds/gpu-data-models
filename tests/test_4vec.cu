#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <vector>
#include <iostream>
#include <random>
#include <chrono>

#include "TCudaVector.h"

 struct FourVector{
  double _px,_py,_pz,_E;
    int _index;
    
};


//we might need to come back on this later on...
__device__ FourVector *fv;

std::vector<FourVector>GenerateRandomTrack(int _size,double mass){
    std::vector<FourVector>track_container;
    std::mt19937_64 rng;
    // initialize the random number generator with time-dependent seed
    uint64_t timeSeed = std::chrono::high_resolution_clock::now().time_since_epoch().count();
    std::seed_seq ss{uint32_t(timeSeed & 0xffffffff), uint32_t(timeSeed>>32)};
    rng.seed(ss);
    // initialize a uniform distribution between 0 and 1
    std::uniform_real_distribution<double> unif(0, 1);
    // ready to generate random numbers
    
    for (int i = 0; i < _size; i++)
    {
        FourVector _temp_track;
        double rand1 = unif(rng);
        double rand2= unif(rng);
        double rand3 = unif(rng);
        
        _temp_track._px = rand1;
        _temp_track._py = rand2;
        _temp_track._pz = rand3;
        _temp_track._E = sqrt(rand1*rand1+rand2*rand2+rand3*rand3+mass*mass);
        _temp_track._index = i;
        track_container.push_back(_temp_track);

    }
     return track_container;     
}

std::vector<double4>GenerateRandomTracks(int _size,double mass){
    std::vector<double4>track_container;
    std::mt19937_64 rng;
    // initialize the random number generator with time-dependent seed
    uint64_t timeSeed = std::chrono::high_resolution_clock::now().time_since_epoch().count();
    std::seed_seq ss{uint32_t(timeSeed & 0xffffffff), uint32_t(timeSeed>>32)};
    rng.seed(ss);
    // initialize a uniform distribution between 0 and 1
    std::uniform_real_distribution<double> unif(0, 1);
    // ready to generate random numbers
    
    for (int i = 0; i < _size; i++)
    {
        double4 _temp_track;
        double rand1 = unif(rng);
        double rand2= unif(rng);
        double rand3 = unif(rng);
        
        _temp_track.x = rand1;
        _temp_track.y = rand2;
        _temp_track.z = rand3;
        _temp_track.w = sqrt(rand1*rand1+rand2*rand2+rand3*rand3+mass*mass);
        track_container.push_back(_temp_track);

    }
     return track_container;     
}


__global__ void testVector(FourVector* in, FourVector* out,int* index_ptr,int _size){
 
    for(int id= blockIdx.x * blockDim.x + threadIdx.x; 
        id<_size;
        id +=blockDim.x * gridDim.x){
        index_ptr[id] = id;
        printf("Device count %d %d \n",id,index_ptr[id]);
       // (out[id])._px;
        out[id]._px = in[id]._px*2.;
        out[id]._py = in[id]._px*2.;
        out[id]._pz = in[id]._pz*2 ;
        out[id]._E = in[id]._E;
        out[id]._index = in[id]._index;
    }
}

__global__ void testVector3(double4* in, double4* out,int* index_ptr,int _size){
 
    for(int id= blockIdx.x * blockDim.x + threadIdx.x; 
        id<_size;
        id +=blockDim.x * gridDim.x){
        index_ptr[id] = id;
       // printf("Device count %d %d \n",id,index_ptr[id]);
       // (out[id])._px;
         out[id].x = in[id].x*2.;
         out[id].y = in[id].x*2.;
         out[id].z = in[id].z*2 ;
         out[id].w = in[id].w;

    }
}

__global__ void testVector2(int* index_ptr,int _size){
        for(int id= blockIdx.x * blockDim.x + threadIdx.x; 
        id<_size;
        id +=blockDim.x * gridDim.x){
        index_ptr[id] = id;
        printf("Device count %d %d \n",id,index_ptr[id]);
        }
}

int RunCudaVectorTypes(){
    
    int numThreads = 1;
    dim3 threadsPerBlock( 1024, 1, 1);
    
    dim3 numberofBlocks( (numThreads + threadsPerBlock.x-1)/threadsPerBlock.x,1,1);   
    
    auto trajectory = GenerateRandomTracks(100,134.34);
    std::cout<<"size of the structure "<<sizeof(double4)<<std::endl;    
    //can I use this into a CudaVector
    //we can offload it into the Cuda friendly stuff..
    CudaVector<double4> trial(trajectory);
    std::cout<<trial[0].x<<std::endl;
    auto dev_trial = trial.ReturnDeviceVector();
    size_t f_size = sizeof(double4);//(4*sizeof(double)+sizeof(int));
    double4* ptr_trial;
    hipMallocManaged((void**)&ptr_trial,100*f_size);
    ptr_trial = thrust::raw_pointer_cast( dev_trial.data() );
    
    
    double4 *ptr_out;
    hipError_t ret = hipMallocManaged((void**)&ptr_out, 100*f_size);
    
    if (ret != hipSuccess) {
    std::cout << hipGetErrorString(ret) << std::endl;
    return 1;}
    
    int *index_ptr;
    hipMallocManaged(&index_ptr,100*sizeof(int));
       
    testVector3<<<1,100>>>(ptr_trial,ptr_out,index_ptr,100);   
    
   // testVector2<<<1,100>>>(index_ptr,100); 
    double4 *host_out = new double4[100];
    
    int *index_out = new int[100];
    
    hipMemcpy(host_out,ptr_out,100*f_size,hipMemcpyDeviceToHost);
    hipMemcpy(index_out,index_ptr,100*sizeof(int),hipMemcpyDeviceToHost);
    
        
    for(int i = 0;i<10;i++){
        std::cout<<index_out[i]<<std::endl;
        std::cout<<host_out[i].x<<std::endl;
        //std::cout<<host_out[i]._px<<" "<<host_out[i]._index<<" "<<trial[i]._px<<
     //   " "<<index_out[i]<<std::endl;   
    }
    free(host_out);
    hipFree(ptr_out);
    return 0;
}

int RunCPPStructTypes(){
    
    int numThreads = 1;
    dim3 threadsPerBlock( 1024, 1, 1);
    
    dim3 numberofBlocks( (numThreads + threadsPerBlock.x-1)/threadsPerBlock.x,1,1);   
    
    auto trajectory = GenerateRandomTrack(100,134.34);
    std::cout<<"size of the structure "<<sizeof(FourVector)<<std::endl;    
    //can I use this into a CudaVector
    //we can offload it into the Cuda friendly stuff..
    CudaVector<FourVector> trial(trajectory);
    std::cout<<trial[0]._px<<std::endl;
    auto dev_trial = trial.ReturnDeviceVector();
    size_t f_size = sizeof(FourVector);//(4*sizeof(double)+sizeof(int));
    FourVector* ptr_trial;
    hipMallocManaged((void**)&ptr_trial,100*f_size);
    ptr_trial = thrust::raw_pointer_cast( dev_trial.data() );
    
    
    FourVector *ptr_out;
    hipError_t ret = hipMallocManaged((void**)&ptr_out, 100*f_size);
    
    if (ret != hipSuccess) {
    std::cout << hipGetErrorString(ret) << std::endl;
    return 1;}
    
    int *index_ptr;
    hipMallocManaged(&index_ptr,100*sizeof(int));
       
    testVector<<<1,100>>>(ptr_trial,ptr_out,index_ptr,100);   
    
   // testVector2<<<1,100>>>(index_ptr,100); 
    FourVector *host_out = new FourVector[100];
    
    int *index_out = new int[100];
    
    hipMemcpy(host_out,ptr_out,100*f_size,hipMemcpyDeviceToHost);
    hipMemcpy(index_out,index_ptr,100*sizeof(int),hipMemcpyDeviceToHost);
    
        
    for(int i = 0;i<10;i++){
        std::cout<<index_out[i]<<std::endl;
        std::cout<<host_out[i]._px<<" "<<host_out[i]._index<<std::endl;
  
    }
    free(host_out);
    hipFree(ptr_out);
    return 0;
}

int main(int argc,char* argv[]){
    
    //RunCudaVectorTypes();
    RunCPPStructTypes();
}
